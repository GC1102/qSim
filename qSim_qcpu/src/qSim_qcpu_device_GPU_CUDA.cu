#include "hip/hip_runtime.h"
/*
 * qSim_qcup_device_GPU_CUDA.cu
 *
 * --------------------------------------------------------------------------
 * Copyright (C) 2022 Gianni Casonato
 *
 * This program is free software: you can redistribute it and/or modify
 * it under the terms of the GNU General Public License as published by
 * the Free Software Foundation, either version 3 of the License.
 *
 * This program is distributed in the hope that it will be useful,
 * but WITHOUT ANY WARRANTY; without even the implied warranty of
 * MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
 * GNU General Public License for more details.
 *
 * You should have received a copy of the GNU General Public License
 * along with this program.  If not, see <http://www.gnu.org/licenses/>.
 * --------------------------------------------------------------------------
 *
 *  Created on: May 4, 2022
 *      Author: gianni
 *
 * Q-CPU support module, providing functions for CUDA based transformations handling,
 * applying a function based approach, and supporting:
 * - single or repeated function block
 * - function block "gap filling" before and after
 *
 * Implemented functions:
 * - basic transformations (I, X, H, CX)
 * - extended transformations (SWAP, Toffoli, Fredkin, QFT) -> TO BE DONE
 * - custom transformations (look-up-table based) -> TO BE DONE
 *
 *  Version History:
 *
 *  Ver   Date       Change
 *  --------------------------------------------------------------------------
 *  1.0   May-2022   Module creation.
 *  1.1   Nov-2022   Instruction set limitation to 1 and 2 qubit gates.
 *                   Transformed to class.
 *                   Defined and handled a function arguments structure.
 *                   Code clean-up.
 *                   Module renamed to qSim_qcup_device_GPU_CUDA.
 *
 *  -------------------------------------------------------------------------- 
 */


#include <stdio.h>
#include <math.h>

#include "qSim_qcpu_device_function_exec.h"
#include "qSim_qcpu_device_GPU_CUDA.h"


#define MIN(x, y) (((x) < (y)) ? (x) : (y))

// --------------------------------------------------------------
//Device 0: "NVIDIA GeForce GT 1030"
//  CUDA Driver Version / Runtime Version          11.4 / 9.2
//  CUDA Capability Major/Minor version number:    6.1
//  Total amount of global memory:                 2001 MBytes (2098135040 bytes)
//  Total amount of constant memory:               65536 bytes
//  Total amount of shared memory per block:       49152 bytes
//  Total number of registers available per block: 65536
//  Warp size:                                     32
//  Maximum number of threads per multiprocessor:  2048
//  Maximum number of threads per block:           1024
//  Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
//  Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)
// --------------------------------------------------------------

// --------------------------------------------------------------
//Device 0: "GRID T4-1Q"
//  CUDA Driver Version / Runtime Version          11.4 / 10.2
//  CUDA Capability Major/Minor version number:    7.5
//  Total amount of global memory:                 1024 MBytes (1073741824 bytes)
//  Total amount of constant memory:               65536 bytes
//  Total amount of shared memory per block:       49152 bytes
//  Total number of registers available per block: 65536
//  Warp size:                                     32
//  Maximum number of threads per multiprocessor:  1024
//  Maximum number of threads per block:           1024
//  Max dimension size of a thread block (x,y,z): (1024, 1024, 64)
//  Max dimension size of a grid size    (x,y,z): (2147483647, 65535, 65535)
// --------------------------------------------------------------

#ifdef __CUDA_DYNPAR__

#define THREADS_PER_BLOCK 32 // best trade-off value for DP kernel

#else

#define THREADS_PER_BLOCK 32 // best trade-off value for SK kernel (similar to 64 or 128)

#endif

// --------------------------------
// kernel entry point functions
// --------------------------------

#ifdef __CUDA_DYNPAR__

// --------------------------------
// multi kernel case

__global__
void kernel_prod_ki(QDEV_ST_VAL_TYPE *x, int idx, int k_start, int k_stop, int k_step,
		            QASM_F_TYPE* d_ftype_cuda_vec, int* d_fn_cuda_vec,
					QDEV_F_ARGS_TYPE* d_fargs_cuda_vec, int tot_f,
					int /*ftype*/, int fn, int fform, int gapn, int futype, int fun, int fuform,
					double* d_y_real, double* d_y_img) {
	// dynamic parallelism case - child kernel
	int jdx = blockIdx.x * blockDim.x + threadIdx.x + k_start; // 1D vector: only x-dimension used
//	printf("ki_dp - level 2...idx: %d  jdx:%d\n", idx, jdx);

	// check for calculation limits considering LSQ gap fillers generated zeroes
	if ((jdx % k_step == idx % k_step) && (jdx < k_stop+1)) {
		// calculate current coefficient
		QDEV_ST_VAL_TYPE k = hipCmul(x[idx], f_dev_qn_exec(idx, jdx, d_ftype_cuda_vec, d_fn_cuda_vec,
				                                          fn, fform, gapn, futype, fun, fuform, d_fargs_cuda_vec, tot_f));

		// use atomic add on real and image parts separately
		atomicAdd(&(d_y_real[idx]), (double)k.x);
		atomicAdd(&(d_y_img[idx]), (double)k.y);
//		printf("fxi_dp - k: %f %f  ---  d_y: %f %f\n", k.x, k.y, d_y_real[i], d_y_img[i]);
	}
//	printf("ki...%d %d -> %f %f\n", i, idx, k->x, k->y);
}

__global__
void kernel_prod_fxi_dp(QDEV_ST_VAL_TYPE *x, QDEV_ST_VAL_TYPE *y, int N,
		                QASM_F_TYPE* d_ftype_cuda_vec, int* d_fn_cuda_vec,
		                QDEV_F_ARGS_TYPE* d_fargs_cuda_vec, int tot_f, int max_block_size, int block_inner_gap_size,
						int ftype, int fn, int fform, int gapn, int futype, int fun, int fuform,
						double* d_y_real, double* d_y_img) {
	// dynamic parallelism case - parent kernel
	int idx = blockIdx.x * blockDim.x + threadIdx.x; // 1D vector: only x-dimension used
//	printf("fxi_dp - level 1...idx: %d\n", idx);

	// combine all i-th row with x elements for y i-th result
	if (idx < N) {
		// define current calculation limits considering LSQ & MSQ gap fillers generated zeroes
	    int k_step = block_inner_gap_size;
		int k_start = max(0, (idx/max_block_size)*max_block_size);
		int k_stop = min(N-1, k_start+max_block_size-1);
		int k_N = k_stop - k_start + 1;
		int nblocks = (k_N+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
		int nthreads = MIN(k_N, THREADS_PER_BLOCK);
		kernel_prod_ki<<<nblocks, nthreads>>>(x, idx, k_start, k_stop, k_step,
				                              d_ftype_cuda_vec, d_fn_cuda_vec, d_fargs_cuda_vec, tot_f,
											  ftype, fn, fform, gapn, futype, fun, fuform, d_y_real, d_y_img);
		hipDeviceSynchronize(); // to sync children kernels - NEEDED!
		y[idx] = QDEV_ST_MAKE_VAL(d_y_real[idx], d_y_img[idx]);
//		printf("fxi_dp...%d -> %f %f\n", idx, y[idx].x, y[idx].y);
	}
}

#else

// --------------------------------
// single kernel case

__global__
void kernel_prod_fxi_sk(QDEV_ST_VAL_TYPE *x, QDEV_ST_VAL_TYPE *y, int N,
						QASM_F_TYPE* d_ftype_cuda_vec, int* d_fn_cuda_vec,
						QDEV_F_ARGS_TYPE* d_fargs_cuda_vec, int tot_f, int max_block_size, int block_inner_gap_size,
						int /*ftype*/, int fn, int fform, int gapn, int futype, int fun, int fuform) {
	// single kernel case
	int idx = blockIdx.x * blockDim.x + threadIdx.x; // 1D vector: only x-dimension used
//	printf("fxi_sk...idx: %d\n", idx);

	// combine all i-th row with x elements for y i-th result
	if (idx < N) {
		y[idx] = QDEV_ST_MAKE_VAL(0.0, 0.0);

		// define current calculation limits considering LSQ & MSQ gap fillers generated zeroes
	    int k_step = block_inner_gap_size;
		int k_start = max(0, (idx/max_block_size)*max_block_size);
		int k_stop = min(N-1, k_start+max_block_size-1);
//		printf("fxi_sk...idx: %d  N: %d  k_step: %d  k_start: %d  k_stop: %d\n", idx, N, k_step, k_start, k_stop);
		
		for (int k=k_start; k<k_stop+1; k++) {
			// check for calculation limits considering LSQ gap fillers generated zeroes
			if (k % k_step == idx % k_step) {
//				printf("...found idx: %d  k: %d", idx, k);
				y[idx] = hipCadd(y[idx], hipCmul(x[k], f_dev_qn_exec(idx, k, d_ftype_cuda_vec, d_fn_cuda_vec,
										       fn, fform, gapn, futype, fun, fuform, d_fargs_cuda_vec, tot_f)));
			}
//			printf("\n");
		}
//		printf("fxi_sk...%d -> %f %f\n", idx, y[idx].x, y[idx].y);
	}
}

#endif

// --------------------------------------------------------
// class methods
// --------------------------------------------------------

#define dev_qreg_MAX_N 20 			// max qureg size supported
#define CUDA_TOT_F dev_qreg_MAX_N  // bounded by max qureg size

// constructor & destructor
qSim_qcpu_device::qSim_qcpu_device() {
    // allocate function host vectors
    m_ftype_vec = (QASM_F_TYPE*)malloc(CUDA_TOT_F*sizeof(QASM_F_TYPE));
    m_fsize_vec = (int*)malloc(CUDA_TOT_F*sizeof(int));
    m_fargs_vec = (QDEV_F_ARGS_TYPE*)malloc(CUDA_TOT_F*sizeof(QDEV_F_ARGS_TYPE));

    // allocate function param vectors
	hipMalloc((void**)&d_ftype_cuda_vec, CUDA_TOT_F*sizeof(QASM_F_TYPE));
	qSim_qcpu_device::checkCUDAError("hipMalloc");
	hipMalloc((void**)&d_fsize_cuda_vec, CUDA_TOT_F*sizeof(int));
	qSim_qcpu_device::checkCUDAError("hipMalloc");
	hipMalloc((void**)&d_fargs_cuda_vec, CUDA_TOT_F*sizeof(QDEV_F_ARGS_TYPE));
	qSim_qcpu_device::checkCUDAError("hipMalloc");

#ifdef __CUDA_DYNPAR__
	// DP case specific part
	int qreg_max_stn = pow(2, dev_qreg_MAX_N);
	hipMalloc(&d_y_real, qreg_max_stn*sizeof(double));
	qSim_qcpu_device::checkCUDAError("hipMalloc");
	hipMalloc(&d_y_img, qreg_max_stn*sizeof(double));
	qSim_qcpu_device::checkCUDAError("hipMalloc");

	hipMemset(d_y_real, 0, qreg_max_stn*sizeof(double));
	qSim_qcpu_device::checkCUDAError("hipMemset");
	hipMemset(d_y_img, 0, qreg_max_stn*sizeof(double));
	qSim_qcpu_device::checkCUDAError("hipMemset");
	hipDeviceSynchronize();	
#endif
}

qSim_qcpu_device::~qSim_qcpu_device() {
	// release function CUDA vectors
	hipFree(d_ftype_cuda_vec);
	hipFree(d_fsize_cuda_vec);
	hipFree(d_fargs_cuda_vec);
	qSim_qcpu_device::checkCUDAError("hipFree");

#ifdef __CUDA_DYNPAR__
	// DP case specific part
	hipFree(d_y_real);
	hipFree(d_y_img);
	qSim_qcpu_device::checkCUDAError("hipFree");
#endif

	// release function host vectors
	free(m_ftype_vec);
	free(m_fsize_vec);
	free(m_fargs_vec);
}

// ---------------------------------------------------------
// instructions execution - qureg transformations
// ---------------------------------------------------------

// => 1-qubit gate functions
int qSim_qcpu_device::dev_qreg_apply_function_gate_1qubit(QDEV_ST_VAL_TYPE*d_x, QDEV_ST_VAL_TYPE*d_y, int d_N,
														 QASM_F_TYPE ftype, int frep, int flsq,
														 QREG_F_ARGS_TYPE* fargs, bool verbose) {
	// handle 1-qubit gate application to given qureg data
	if (verbose) {
		printf("applying 1-qubit gate function...\n");
		printf("d_N: %d - ftype: %d - frep: %d - flsq: %d - fargs size: %lu\n",
				d_N, ftype, frep, flsq, fargs->size());
	}

	// convert fargs to CUDA device pointer array
	QDEV_F_ARGS_TYPE dev_fargs = qSim_qcpu_device::fargs_to_dev_ptr_array(*fargs);
	if (verbose)
		printf("dev_fargs...argc: %d - argv: %g\n", dev_fargs.argc, dev_fargs.argv);

	// perform function aggregation for gap filling w.r.t overall qureg size
	int fn = 1; // fixed for 1-qubit gate
	int fsize = 2; // fixed for 1-qubit gate
	int tot_f = f_dev_gap_filling(d_N, ftype, fsize, frep, flsq, dev_fargs,
								   m_ftype_vec, m_fsize_vec, m_fargs_vec, verbose);
	if (tot_f < 1) {
		printf("dev_qreg_apply_function_gate_1qubit: 0 functions returned by gap filling - error!!\n");
		return QDEV_RES_ERROR; // return error
	}
	int max_block_size = powf(2, fn*frep + flsq);
	int block_inner_gap_size = powf(2, flsq);
	if (verbose) {
		printf("dev_qreg_apply_function_gate_1qubit: gap filling tot_f: %d  max_block_size: %d  block_inner_gap_size: %d\n",
				tot_f, max_block_size, block_inner_gap_size);
	}

	// store function type vectors into CUDA device memory objects for use in kernels
	dev_vec_host2device((void**)&d_ftype_cuda_vec, m_ftype_vec, tot_f, sizeof(QASM_F_TYPE));
	dev_vec_host2device((void**)&d_fsize_cuda_vec, m_fsize_vec, tot_f, sizeof(int));
	if (verbose)
		printf("ftype & fsize prepared...\n");

	// convert function args vector to CUDA memory objects
	dev_args_host2device(&d_fargs_cuda_vec, m_fargs_vec, tot_f);
	if (verbose) {
		printf("fargs prepared...\n");
//		for (int jj=0; jj<tot_f; jj++) {
//			printf("#%d - d_fargs_cuda: (%d %f)\n", jj, d_fargs_cuda_vec[jj].argc, d_fargs_cuda_vec[jj].argv);
//		}
	}

	// perform kernel function on N elements
	int nblocks = (d_N+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
	int nthreads = MIN(d_N, THREADS_PER_BLOCK);
	if (verbose)
		printf("nblocks: %d  nthreads: %d\n\n", nblocks, nthreads);

#ifdef __CUDA_DYNPAR__
	// dynamic parallelism mode
	if (verbose)
		printf("calling kernel...DP\n\n");
	kernel_prod_fxi_dp<<<nblocks, nthreads>>>(d_x, d_y, d_N, d_ftype_cuda_vec, d_fsize_cuda_vec,
			                                  d_fargs_cuda_vec, tot_f, max_block_size, block_inner_gap_size,
											  ftype, fn, 0, 0, 0, QASM_F_TYPE_NULL, 0, // form n.a. here
											  d_y_real, d_y_img);
	qSim_qcpu_device::checkCUDAError("kernel_prod_fxi_dp");

#else
	// single kernel mode
	if (verbose)
		printf("calling kernel...SK\n\n");
	kernel_prod_fxi_sk<<<nblocks, nthreads>>>(d_x, d_y, d_N, d_ftype_cuda_vec, d_fsize_cuda_vec,
											  d_fargs_cuda_vec, tot_f, max_block_size, block_inner_gap_size,
											  ftype, fn, 0, 0, 0, QASM_F_TYPE_NULL, 0); // form n.a. here
	qSim_qcpu_device::checkCUDAError("kernel_prod_fxi_sk");

#endif

	// wait for all kernel instances to complete
	hipDeviceSynchronize();

	if (verbose)
		printf("qreg_apply_function done\n");

	return QDEV_RES_OK;
}

// --------------------------------

// => 2-qubit gate functions
int qSim_qcpu_device::dev_qreg_apply_function_gate_2qubit(QDEV_ST_VAL_TYPE*d_x, QDEV_ST_VAL_TYPE*d_y, int d_N,
														 QASM_F_TYPE ftype, int frep, int flsq, int fform, int futype,
														 QREG_F_ARGS_TYPE* fuargs, bool verbose) {
	// handle 2-qubit gate application to given qureg data
	if (verbose) {
		printf("applying 2-qubit gate function...\n");
		printf("d_N: %d - ftype: %d - frep: %d - flsq: %d - fform: %d - futype: %d - fargs size: %lu\n",
				d_N, ftype, frep, flsq, fform, futype, fuargs->size());
	}

	// convert fargs to CUDA device pointer array
	QDEV_F_ARGS_TYPE dev_fuargs = qSim_qcpu_device::fargs_to_dev_ptr_array(*fuargs);
	if (verbose)
		printf("dev_fuargs...argc: %d - argv: %g\n", dev_fuargs.argc, dev_fuargs.argv);

	// perform function aggregation for gap filling w.r.t overall qureg size
	int fn = 2; // fixed for 1-qubit gate
	int fsize = 4; // fixed for 1-qubit gate
	int fuform = 0; // fixed for 2-qubit case
	int tot_f = f_dev_gap_filling(d_N, ftype, fsize, frep, flsq, dev_fuargs,
								   m_ftype_vec, m_fsize_vec, m_fargs_vec, verbose);
	if (tot_f < 1) {
		printf("dev_qreg_apply_function_gate_1qubit: 0 functions returned by gap filling - error!!");
		return QDEV_RES_ERROR; // return error
	}
	int max_block_size = powf(2, fn*frep + flsq);
	int block_inner_gap_size = powf(2, flsq);
	if (verbose) {
		printf("dev_qreg_apply_function_gate_1qubit: gap filling tot_f: %d  max_block_size: %d  block_inner_gap_size: %d\n",
				tot_f, max_block_size, block_inner_gap_size);
	}

	// store function type vectors into CUDA device memory objects for use in kernels
	dev_vec_host2device((void**)&d_ftype_cuda_vec, m_ftype_vec, tot_f, sizeof(QASM_F_TYPE));
	dev_vec_host2device((void**)&d_fsize_cuda_vec, m_fsize_vec, tot_f, sizeof(int));
	if (verbose)
		printf("ftype & fsize prepared...\n");

	// convert function args vector to CUDA memory objects
	dev_args_host2device(&d_fargs_cuda_vec, m_fargs_vec, tot_f);
	if (verbose) {
		printf("fargs prepared...\n");
		// for (int jj=0; jj<tot_f; jj++) {
			// printf("#%d - d_fargs_cuda: (%d %f)\n", jj, d_fargs_cuda_vec[jj].argc, d_fargs_cuda_vec[jj].argv);
		// }
	}

	// perform kernel function on N elements
	int nblocks = (d_N+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
	int nthreads = MIN(d_N, THREADS_PER_BLOCK);
	if (verbose)
		printf("nblocks: %d  nthreads: %d\n\n", nblocks, nthreads);

#ifdef __CUDA_DYNPAR__
	// dynamic parallelism mode
	if (verbose)
		printf("calling kernel...DP\n\n");
	kernel_prod_fxi_dp<<<nblocks, nthreads>>>(d_x, d_y, d_N, d_ftype_cuda_vec, d_fsize_cuda_vec,
			                                  d_fargs_cuda_vec, tot_f, max_block_size, block_inner_gap_size,
											  ftype, fn, fform, 0, futype, 1, fuform,
											  d_y_real, d_y_img);
	qSim_qcpu_device::checkCUDAError("kernel_prod_fxi_dp");

#else
	// single kernel mode
	if (verbose)
		printf("calling kernel...SK\n\n");
	kernel_prod_fxi_sk<<<nblocks, nthreads>>>(d_x, d_y, d_N, d_ftype_cuda_vec, d_fsize_cuda_vec,
											  d_fargs_cuda_vec, tot_f, max_block_size, block_inner_gap_size,
											  ftype, fn, fform, 0, futype, 1, fuform);
	qSim_qcpu_device::checkCUDAError("kernel_prod_fxi_sk");

#endif

	// wait for all kernel instances to complete
	hipDeviceSynchronize();

	if (verbose)
		printf("qreg_apply_function done\n");

	return QDEV_RES_OK;
}

// --------------------------------

// => n-qubit gate functions
int qSim_qcpu_device::dev_qreg_apply_function_controlled_gate_nqubit(QDEV_ST_VAL_TYPE*d_x, QDEV_ST_VAL_TYPE*d_y, int d_N,
																	 QREG_F_TYPE ftype, int fsize, int frep, int flsq, int fform,
																	 int fgapn, int futype, int fun, int fuform,
																	 QREG_F_ARGS_TYPE* fuargs, bool verbose) {
	// handle 2-qubit gate application to given qureg data
	if (verbose) {
		printf("applying n-qubit gate function...\n");
		printf("d_N: %d - ftype: %d - fsize: %d - frep: %d - flsq: %d - fform: %d - futype: %d - fargs size: %lu\n",
				d_N, ftype, fsize, frep, flsq, fform, futype, fuargs->size());
	}

	// convert fargs to CUDA pointer array
	QDEV_F_ARGS_TYPE dev_fargs = qSim_qcpu_device::fargs_to_dev_ptr_array(*fuargs);

	// perform function aggregation for gap filling w.r.t overall qureg size
	int fn = log2(fsize); // function size in qubits
	int tot_f = f_dev_gap_filling(d_N, ftype, fsize, frep, flsq, dev_fargs,
								   m_ftype_vec, m_fsize_vec, m_fargs_vec, verbose);
	if (tot_f < 1) {
		printf("dev_qreg_apply_function_gate_1qubit: 0 functions returned by gap filling - error!!");
		return QDEV_RES_ERROR; // return error
	}
	int max_block_size = powf(2, fn*frep + flsq);
	int block_inner_gap_size = powf(2, flsq);
	if (verbose) {
		printf("dev_qreg_apply_function_gate_1qubit: gap filling tot_f: %d  max_block_size: %d  block_inner_gap_size: %d\n",
				tot_f, max_block_size, block_inner_gap_size);
	}

	// store function type vectors into CUDA device memory objects for use in kernels
	dev_vec_host2device((void**)&d_ftype_cuda_vec, m_ftype_vec, tot_f, sizeof(QASM_F_TYPE));
	dev_vec_host2device((void**)&d_fsize_cuda_vec, m_fsize_vec, tot_f, sizeof(int));
	if (verbose)
		printf("ftype & fsize prepared...\n");

	// convert function args vector to CUDA memory objects
	dev_args_host2device(&d_fargs_cuda_vec, m_fargs_vec, tot_f);
	if (verbose) {
		printf("fargs prepared...\n");
		// for (int jj=0; jj<tot_f; jj++) {
			// printf("#%d - d_fargs_cuda: (%d %f)\n", jj, d_fargs_cuda_vec[jj].argc, d_fargs_cuda_vec[jj].argv);
		// }
	}

	// perform kernel function on N elements
	int nblocks = (d_N+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
	int nthreads = MIN(d_N, THREADS_PER_BLOCK);
	if (verbose)
		printf("nblocks: %d  nthreads: %d\n\n", nblocks, nthreads);

#ifdef __CUDA_DYNPAR__
	// dynamic parallelism mode
	if (verbose)
		printf("calling kernel...DP\n\n");
	kernel_prod_fxi_dp<<<nblocks, nthreads>>>(d_x, d_y, d_N, d_ftype_cuda_vec, d_fsize_cuda_vec,
			                                  d_fargs_cuda_vec, tot_f, max_block_size, block_inner_gap_size,
											  ftype, fn, fform, fgapn, futype, fun, fuform,
											  d_y_real, d_y_img);
	qSim_qcpu_device::checkCUDAError("kernel_prod_fxi_dp");

#else
	// single kernel mode
	if (verbose)
		printf("calling kernel...SK\n\n");
	kernel_prod_fxi_sk<<<nblocks, nthreads>>>(d_x, d_y, d_N, d_ftype_cuda_vec, d_fsize_cuda_vec,
											  d_fargs_cuda_vec, tot_f, max_block_size, block_inner_gap_size,
											  ftype, fn, fform, fgapn, futype, fun, fuform);
	qSim_qcpu_device::checkCUDAError("kernel_prod_fxi_sk");

#endif

	// wait for all kernel instances to complete
	hipDeviceSynchronize();

	if (verbose)
		printf("qreg_apply_function done\n");

	return QDEV_RES_OK;
}

// ---------------------------------------------------------
// instructions execution - qureg state handling
// ---------------------------------------------------------

// => qureg state setup - kernel function
__global__
void kernel_set_state(QDEV_ST_VAL_TYPE *x, int N, int st_val) {
	int idx = blockIdx.x * blockDim.x + threadIdx.x; // 1D vector: only x-dimension used
//	printf("N: %d  idx: %d\n", rN, idx);
	if (idx < N)
		if (idx != st_val)
			x[idx] = QDEV_ST_MAKE_VAL(0.0, 0.0);
		else
			x[idx] = QDEV_ST_MAKE_VAL(1.0, 0.0);
}

// --------------------------------

// => qureg state value set
void qSim_qcpu_device::dev_qreg_set_state(QDEV_ST_VAL_TYPE*d_x, int N, int st_val, bool verbose) {
	// perform kernel function on N elements
	int nblocks = (N+THREADS_PER_BLOCK-1)/THREADS_PER_BLOCK;
	int nthreads = MIN(N, THREADS_PER_BLOCK);
	if (verbose) {
		printf("CUDA - qreg_set_state...st_val: %d\n", st_val);
		printf("nblocks: %d  nthreads: %d\n\n", nblocks, nthreads);
	}

	// call CUDA kernel functions
	kernel_set_state<<<nblocks, nthreads>>>(d_x, N, st_val);
	qSim_qcpu_device::checkCUDAError("kernel_set_state");

	// wait for all kernel instances to complete
	hipDeviceSynchronize();
}

// ---------------------------------------------------------
// static helper host <--> device conversion methods
// ---------------------------------------------------------

void qSim_qcpu_device::dev_qreg_host2device(QDEV_ST_VAL_TYPE** d_x, QDEV_ST_VAL_TYPE* x, int N) {
	// allocate and setup device memory with given host one
	hipMalloc((void**)d_x, N*sizeof(QDEV_ST_VAL_TYPE));
	checkCUDAError("hipMalloc");
	hipMemcpy((*d_x), x, N*sizeof(QDEV_ST_VAL_TYPE), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy");
//	hipDeviceSynchronize();	-> not needed after a hipMemcpy (it is synchronous)
}

void qSim_qcpu_device::dev_qreg_device2host(QDEV_ST_VAL_TYPE* x, QDEV_ST_VAL_TYPE* d_x, int N) {
	// allocate and setup device memory with given host one
	hipMemcpy(x, d_x, N*sizeof(QDEV_ST_VAL_TYPE), hipMemcpyDeviceToHost);
	checkCUDAError("hipMemcpy");
//	hipDeviceSynchronize();	-> not needed after a hipMemcpy (it is synchronous)
}

void qSim_qcpu_device::dev_qreg_host2device_align(QDEV_ST_VAL_TYPE* d_x, QDEV_ST_VAL_TYPE* x, int N) {
	// device memory alignment with given host one - no allocation
	hipMemcpy(d_x, x, N*sizeof(QDEV_ST_VAL_TYPE), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy");
//	hipDeviceSynchronize();	-> not needed after a hipMemcpy (it is synchronous)
}

void qSim_qcpu_device::dev_vec_host2device(void** d_x, void* x, int n, int size) {
	// allocate and setup unified memory with given one
	hipMemcpy((*d_x), x, n*size, hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy");
//	hipDeviceSynchronize();	-> not needed after a hipMemcpy (it is synchronous)
}

void qSim_qcpu_device::dev_args_host2device(QDEV_F_ARGS_TYPE** d_fargs, QDEV_F_ARGS_TYPE* fargs, int tot_f) {
	// allocate and setup unified memory with given one - iterating over function arguments
	hipMemcpy((*d_fargs), fargs, tot_f*sizeof(QDEV_F_ARGS_TYPE), hipMemcpyHostToDevice);
	checkCUDAError("hipMemcpy");
//	hipDeviceSynchronize();	-> not needed after a hipMemcpy (it is synchronous)
}

// helper function for device memory release
void qSim_qcpu_device::dev_qreg_device_release(QDEV_ST_VAL_TYPE* d_x) {
	hipFree(d_x);
	checkCUDAError("hipFree");
}

// helper function for kernel error codes check
void qSim_qcpu_device::checkCUDAError(const char* cmd_msg) {
    hipError_t err = hipGetLastError();
    if (hipSuccess != err) {
    	fprintf(stderr, "CUDA error executing %s: err: %d  msg: %s.\n",
    			cmd_msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

// ---------------------------------

// QASM function arguments conversion into CUDA device structure
QDEV_F_ARGS_TYPE qSim_qcpu_device::fargs_to_dev_ptr_array(QREG_F_ARGS_TYPE fargs) {
	// convert from QASM to CUDA device formats
	QDEV_F_ARGS_TYPE cuda_fargs;
	if (fargs.size() > 0) {
		// some argument found
		cuda_fargs.argc = fargs.size();
		for (unsigned int i=0; i<fargs.size(); i++) {
			if (fargs[i].m_type == qSim_qreg_function_arg::INT) {
				cuda_fargs.argv = (double)fargs[i].m_i;
			}
			else {
				cuda_fargs.argv = fargs[i].m_d;
			}
		}
	}
	else {
		// no arguments
		cuda_fargs.argc = 0;
		cuda_fargs.argv = 0.0;
	}
	return cuda_fargs;
}

// ---------------------------------------------------------
// CUDA device inspection methods
// ---------------------------------------------------------

// => CUDA device count
int qSim_qcpu_device::dev_get_gpu_cuda_count() {
	int nDevices = 0;
#ifndef _WIN32
	// linux case
	hipGetDeviceCount(&nDevices);
	checkCUDAError("hipGetDeviceCount");
#else
	// windows case
	printf("dev_get_gpu_cuda_count non supported by Windows GPU!\n\n");
	nDevices = 1;
#endif
	return nDevices;
}

// => CUDA device info
void qSim_qcpu_device::dev_gpu_cuda_properties_dump() {
#ifndef _WIN32
	// linux case
	int nDevices = dev_get_gpu_cuda_count();
	for (int i = 0; i < nDevices; i++) {
		hipDeviceProp_t prop;
		hipGetDeviceProperties(&prop, i);
		checkCUDAError("hipGetDeviceCount");

		printf("GPU CUDA Device Number: %d\n", i);
		printf("  Device name: %s\n", prop.name);
		printf("  Total Global Memory (GB): %4.3f\n", prop.totalGlobalMem/1e9);
		printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
		printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
		printf("  Peak Memory Bandwidth (GB/s): %f\n",
			   2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
		printf("\n");
	}
	printf("--------------------------------\n\n");
#else
// windows case
	printf("dev_gpu_cuda_properties_dump non supported by Windows GPU!\n\n");
#endif
}

// --------------------------------

